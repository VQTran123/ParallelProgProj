#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


__global__ void merge(int* arraylist, int* temparray, int left, int right, int rank)
{
    //variable
    int mid = (left + right)/2;
    int length = right - left;
    //stream for right array and left array
    hipStream_t startleft, startright;
    
    //start to return if the array is 1 or 0
    if(length < 2){
        return;
    }
    //create block for the left array
    hipStreamCreateWithFlags(&startleft, hipStreamNonBlocking);
    merge<<<1,1,0, startleft>>>(arraylist, temparray, left, mid, rank + 1);
    hipStreamDestroy(startleft);
    
    //create block for the right array
    hipStreamCreateWithFlags(&startright, hipStreamNonBlocking);
    merge<<<1,1,0,startright>>>>(arraylist, temparray, mid, right, rank + 1);
    hipStreamDestroy(startright);

    hipDeviceSynchronize();
    int i;
    int templeft = left;
    int tempmid = mid;
    for(i = left; i < right; i++){
        if(templeft < mid && (tempmid >= right || arraylist[templeft] <= arraylist[tempmid])){
            temparray[i] = arraylist[templeft];
            templeft++;
        }
        else{
            temparray[i] = arraylist[tempmid];
            tempmid++;
        }
    }
    for(i = left; i < right; index++){
        arraylist[i] = temparray[i];
    }
}

extern "C" void sort(int* array, int num){
    int* gpuarray;
    int* temparray;
    int leftnum = 0;
    int rightnum = 0;

    int arraysize = num * sizeof(int);
    //allocating memory for the array
    hipMalloc((void**)&gpuarray, arraysize);
    hipMalloc((void**)&temparray, arraysize);
    hipMemcpy(gpuarray, array, arraysize, hipMemcpyHostToDevice);
    //call the merge function to start sorting
    merge<<<1, 1>>>>(gpuarray, temparray, leftnum, rightnum, 0);
    hipDeviceSynchronize();
    //copy the array in the sorting function back to the main array
    c    //free the array    hipMemcpy(array, gpuarray, arraysize, hipMemcpyDeviceToHost);
    hipFree(temparray);

temparray    gpuarray);
    cudaDevi
ceReset();}

extern "C" void initialize_CUDA(int rank){
    int cudaDeviceCount;
    int cE = hipGetDeviceCount( &cudaDeviceCount);
    if( cE != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
            cE, cudaDeviceCount );
        exit(-1);
    }
    cE = hipGetDeviceCount( &cudaDeviceCount);
    if( cE != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
            rank, (rank % cudaDeviceCount), cE);
        exit(-1);
    }
}